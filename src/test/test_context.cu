#include <thread>
#include "gtest/gtest.h"
#include "singa/utils/singleton.h"
#include "singa/utils/context.h"
#include "singa/utils/cuda_utils.h"

using namespace singa;
using namespace std;

TEST(ContextTest, TestDevice) {
  auto context = Singleton<Context>::Instance();

  auto id = std::this_thread::get_id();
  context->SetupDevice(id, 0);
  auto device_id = context->device_id(id);
  ASSERT_EQ(1,device_id);
}

TEST(ContextTest, TestHandle) {
  auto context = Singleton<Context>::Instance();

  float cpu_ret = 0.0f;
  float gpu_ret = 0.0f;

  float A[12];
  float B[12];

  for(int i = 0; i < 12; i++) {
    A[i]=i-1;
    B[i]=i+1;
  }

  float* A_gpu = NULL;
  float* B_gpu = NULL;
  context->SetupDevice(std::this_thread::get_id(), 0);

  hipMalloc((void**)&A_gpu, 12 * sizeof(float));
  hipMalloc((void**)&B_gpu, 12 * sizeof(float));

  hipMemcpy(A_gpu, A, 12 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, 12 * sizeof(float), hipMemcpyHostToDevice);

  hipblasHandle_t handle = context->cublas_handle(std::this_thread::get_id());

  hipblasSdot(handle, 12, A_gpu, 1, B_gpu, 1, &gpu_ret);

  for(int i = 0; i < 12;++i) {
    cpu_ret += A[i] * B[i];
  }

  ASSERT_EQ(gpu_ret,cpu_ret);

  hipFree(A_gpu);
  hipFree(B_gpu);
}
